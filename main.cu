#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>

using namespace nvcuda;

typedef signed char  ElementAB;
typedef int ElementC;

__global__ void wmma_ker(const ElementAB* a, const ElementAB* b, ElementC* c, const int N) {
    wmma::fragment<wmma::matrix_a, 16, 16, 16, ElementAB, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, ElementAB, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, ElementC> c_frag;

       
    // Initialize the output to zero
    wmma::fill_fragment(c_frag, __float2half(.0f));
    // Load the inputs

    for (auto k = 0; k < N; k += 16) {
        wmma::load_matrix_sync(a_frag, &a[blockIdx.y * N * 16 + k], N);
        wmma::load_matrix_sync(b_frag, &b[k * N + blockIdx.x * 16], N);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
   // Store the output
    wmma::store_matrix_sync(&c[blockIdx.y * N * 16 + blockIdx.x * 16], c_frag, N, wmma::mem_row_major);
}

int main() {

    ElementAB *inputA, *inputB;
    ElementC *outputC;
    const int N = 16;
    const int size = N * N;
    hipMallocManaged(&inputA, size * sizeof(ElementAB));
    hipMallocManaged(&inputB, size * sizeof(ElementAB));
    hipMallocManaged(&outputC, size * sizeof(ElementC));

    for (int i = 0; i < size; i++) {
        inputA[i] = 1;
        inputB[i] = 1;
    }
    
    wmma_ker<<<dim3(N / 16, N / 16), dim3(16, 16)>>>(inputA, inputB, outputC, N);

    hipDeviceSynchronize();

    for (int i = 0; i < size; i++) {
        std::cout << outputC[i];
    }
    std::cout << std::endl;

    hipFree(inputA);
    hipFree(inputB);
    hipFree(outputC);

    return 0;
}
